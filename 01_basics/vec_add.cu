// Adding all the libraries
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <assert.h>


// Cuda Kernel for vector addition
__global__ void vectorAddition(int* a, int* b, int* c, int n) {
    // Calculate thread id
    int thread_id = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Vector boundary guard
    if (thread_id < n) {
        // Each thread adds a single element
        c[thread_id] = a[thread_id] + b[thread_id];
    }
}

// Initialize vector of size n
void matrixInit(int* a, int n) {
    for(int i=0; i<n; i++) {
        a[i] = rand() % 100;
    }
}

// Check add result
void errorCheck(int* a, int* b, int* c, int n) {
    for(int i=0; i<n; i++) {
        assert(c[i] == (a[i] + b[i]));
    }
}

int main() {
    int n = 1 << 16; // size of array/vector

    int *h_a, *h_b, *h_c; // host vector pointers
    int *d_a, *d_b, *d_c; // device vector pointers

    size_t bytes = sizeof(int) * n; // size of array/vector in bytes

    // Allocate memory for host
    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);

    // Allocate device memory
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Initialize vectors
    matrixInit(h_a, n);
    matrixInit(h_b, n);

    // Copy data to GPU
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    // Threadblock size
    int NUM_THREADS = 256;

    // Grid size
    int NUM_BLOCKS = (int)ceil(n / NUM_THREADS);

    // Launch Kernel on default stream w/o shmem
    vectorAddition<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, n);

    // Copy sum vector from device to host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    // Check for errors
    errorCheck(h_a, h_b, h_c, n);

    printf("SUCCESS- Completed\n");

    return 0;
}